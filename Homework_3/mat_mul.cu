#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <random>
#include <chrono>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if(__err != hipSuccess) { \
                fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                                msg, hipGetErrorString(__err), \
                                __FILE__, __LINE__); \
                fprintf(stderr, "***FAILED - ABORTING\n"); \
                exit(1); \
        } \
}while(0)

#define TILE_SIZE 16

__global__ void mat_mul(uint64_t* matA, uint64_t* matB, uint64_t* resC, uint64_t N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;  // Global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Global column index

    uint64_t value = 0;

    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            value += matA[row * N + i] * matB[i * N + col];
        }
        resC[row * N + col] = value;
    }
	
}

void init_mat(uint64_t* mat, uint64_t size){
	srand(766);
	for(int i = 0; i < size; i++)
		mat[i] = (rand() % 1000) + i;
}

int check_compute(uint64_t* A, uint64_t* B, uint64_t* C, uint64_t* d_C, uint64_t N) {
	
	for (int i = 0; i < N; i++) {          // Loop over rows of A
        for (int j = 0; j < N; j++) {      // Loop over columns of B
            for (int k = 0; k < N; k++) {  // Loop over columns of A / rows of B
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
	
	for(int i = 0; i < N ; i++) {
		if (C[i] != d_C[i]){
			return -1;
		}
	}
	return 0;
}

int main(int argc, char* argv[]) {
	hipDeviceReset();

	uint64_t N ;
	if (argc != 2) {
		printf("Usage ./mat_mul <size>\n");
		exit(1);
	}
	else {
		N = (uint64_t)atoi(argv[1]);
	}

	uint64_t *d_A, *d_B, *d_resC;
	uint64_t *h_A, *h_B, *h_C, *h_resC;

	hipMalloc(&d_A, N*N*sizeof(uint64_t));
	hipMalloc(&d_B, N*N*sizeof(uint64_t));
	hipMalloc(&d_resC, N*N*sizeof(uint64_t));

	h_A = (uint64_t*)malloc(N*N*sizeof(uint64_t));
	h_B = (uint64_t*)malloc(N*N*sizeof(uint64_t));
	h_resC = (uint64_t*)malloc(N*N*sizeof(uint64_t));
	h_C = (uint64_t*)malloc(N*N*sizeof(uint64_t));
	memset(h_resC, 0, N*N*sizeof(uint64_t));
	memset(h_C, 0, N*N*sizeof(uint64_t));

	init_mat(h_A, N);
	init_mat(h_B, N);

	hipMemcpy(d_A, h_A, N*N*sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, N*N*sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(d_resC, h_resC, N*N*sizeof(uint64_t), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
	dim3 numBlocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

	auto start = std::chrono::high_resolution_clock::now();
	mat_mul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_resC, N);
	cudaCheckErrors("kernel failed to launch\n");
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> elapsed = end - start;

	hipMemcpy(h_resC, d_resC, N*N*sizeof(uint64_t), hipMemcpyDeviceToHost);
	
	//int ret = check_compute(h_A, h_B, h_C, h_resC, N);
	
	//if(ret == 0)
	printf("%lu, %f, Successful\n", N, elapsed.count());
	//else{
		//printf("Failed\n");
		//return -1;
	//}
	return 0;


}
